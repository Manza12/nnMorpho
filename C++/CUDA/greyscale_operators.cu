#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../greyscale_operators.h"

// Erosion
template <typename scalar>
torch::Tensor erosion(torch::Tensor input,
                      torch::Tensor str_el,
                      torch::Tensor footprint,
                      int origin_x,
                      int origin_y,
                      char border_type,
                      scalar top,
                      scalar bottom,
                      const int block_size_x,
                      const int block_size_y) {

    // Compute output size
    const auto m = input.size(1);
    const auto n = input.size(0);
    const auto p = str_el.size(1);
    const auto q = str_el.size(0);

    // Initialization
    auto options = torch::TensorOptions().device(input.device()).dtype(input.dtype());
    torch::Tensor output_tensor = torch::zeros({n, m}, options);

    // Switch between CPU and GPU
    if (input.is_cuda()) {
        /* GPU */
        // Create accessors
        auto input_accessor = input.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();
        auto str_el_accessor = str_el.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();
        auto footprint_accessor = footprint.packed_accessor32<bool, 2, torch::RestrictPtrTraits>();
        auto output_accessor = output_tensor.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();

        // Block & Grid parameters
        const int grid_x = ((m - 1) / block_size_x) + 1;
        const int grid_y = ((n - 1) / block_size_y) + 1;

        const dim3 block_size(block_size_x, block_size_y, 1);
        const dim3 grid_size(grid_x, grid_y, 1);

        // Launch of the kernel
        erosion_cuda_kernel<<<grid_size, block_size>>>(input_accessor, str_el_accessor, footprint_accessor,
                                                       output_accessor);
    } else {
        /* CPU */
        // Create accessors
        auto input_accessor = input.accessor<scalar, 2>();
        auto str_el_accessor = str_el.accessor<scalar, 2>();
        auto footprint_accessor = footprint.accessor<bool, 2>();
        auto output_accessor = output_tensor.accessor<scalar, 2>();

        scalar value;
        scalar difference;
        int idx_x, idx_y;
        // Computation
        for (int y = 0; y < n; y++) {
            for (int x = 0; x < m; x++) {
                value = top;
                // Compute the value of output[y][x]
                for (int j = 0; j < q; j++) {
                    for (int i = 0; i < p; i++) {
                        if (footprint_accessor[j][i]) {
                            idx_x = x + (i - origin_x);
                            idx_y = y + (j - origin_y);
                            if (0 <= idx_x && idx_x < m && 0 <= idx_y && idx_y < n) {
                                difference = input_accessor[idx_y][idx_x] - str_el_accessor[j][i];
                                if (value > difference) value = difference;
                            } else if (border_type == 'e') {
                                value = bottom;
                                goto end;
                            }
                        }
                    }
                }
                end: output_accessor[y][x] = value;
            }
        }
    }

    return output_tensor;
};

// Dilation
template <typename scalar>
torch::Tensor dilation(torch::Tensor input,
                       torch::Tensor str_el,
                       torch::Tensor footprint,
                       int origin_x,
                       int origin_y,
                       scalar bottom,
                       int block_size_x,
                       int block_size_y) {

    // Compute output size
    const auto m = input.size(1);
    const auto n = input.size(0);
    const auto p = str_el.size(1);
    const auto q = str_el.size(0);

    // Initialization
    auto options = torch::TensorOptions().device(input.device()).dtype(input.dtype());
    torch::Tensor output_tensor = torch::zeros({n, m}, options);

    // Create accessors
    auto input_accessor = input.accessor<scalar, 2>();
    auto str_el_accessor = str_el.accessor<scalar, 2>();
    auto footprint_accessor = footprint.accessor<bool, 2>();
    auto output_accessor = output_tensor.accessor<scalar, 2>();

    scalar value;
    scalar sum;
    int idx_x, idx_y;
    // Computation
    for (int y = 0; y < n; y++) {
        for (int x = 0; x < m; x++) {
            value = bottom;
            // Compute the value of output[y][x]
            for (int j = q-1; j >= 0; j--) {
                for (int i = p-1; i >= 0; i--) {
                    if (footprint_accessor[j][i]) {
                        idx_x = x - (i - origin_x);
                        idx_y = y - (j - origin_y);
                        if (0 <= idx_x && idx_x < m && 0 <= idx_y && idx_y < n) {
                            sum = input_accessor[idx_y][idx_x] + str_el_accessor[j][i];
                            if (value < sum) value = sum;
                        }
                    }
                }
            }
            output_accessor[y][x] = value;
        }
    }

    return output_tensor;
};
