#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "greyscale_operators.cpp"

/* Kernels */
// Erosion
template <typename scalar>
__global__ void erosion_cuda_kernel(
        const torch::PackedTensorAccessor32<scalar, 2, torch::RestrictPtrTraits> input_accessor,
        const torch::PackedTensorAccessor32<scalar, 2, torch::RestrictPtrTraits> str_el_accessor,
        const torch::PackedTensorAccessor32<bool, 2, torch::RestrictPtrTraits> footprint_accessor,
        torch::PackedTensorAccessor32<scalar, 2, torch::RestrictPtrTraits> output_accessor,
        const int origin_x,
        const int origin_y,
        char border_type,
        scalar top,
        scalar bottom) {

    // Sizes
    const auto m = input_accessor.size(1);
    const auto n = input_accessor.size(0);
    const auto p = str_el_accessor.size(1);
    const auto q = str_el_accessor.size(0);

    // Compute thread index corresponding in output tensor
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Declare variables
    scalar value = top;
    scalar difference;
    int idx_x, idx_y;

    // Compute the value of output[y][x]
    if (x < m && y < n) {
        for (int j = 0; j < q; j++) {
            for (int i = 0; i < p; i++) {
                if (footprint_accessor[j][i]) {
                    idx_x = x + (i - origin_x);
                    idx_y = y + (j - origin_y);
                    if (0 <= idx_x && idx_x < m && 0 <= idx_y && idx_y < n) {
                        difference = input_accessor[idx_y][idx_x] - str_el_accessor[j][i];
                        if (value > difference) value = difference;
                    } else if (border_type == 'e') {
                        value = bottom;
                        goto end;
                    }
                }
            }
        }
        end: output_accessor[y][x] = value;
    }
}

// Dilation
template <typename scalar>
__global__ void dilation_cuda_kernel(
        const torch::PackedTensorAccessor32<scalar, 2, torch::RestrictPtrTraits> input_accessor,
        const torch::PackedTensorAccessor32<scalar, 2, torch::RestrictPtrTraits> str_el_accessor,
        const torch::PackedTensorAccessor32<bool, 2, torch::RestrictPtrTraits> footprint_accessor,
        torch::PackedTensorAccessor32<scalar, 2, torch::RestrictPtrTraits> output_accessor,
        const int origin_x,
        const int origin_y,
        const scalar bottom) {

    // Sizes
    const auto m = input_accessor.size(1);
    const auto n = input_accessor.size(0);
    const auto p = str_el_accessor.size(1);
    const auto q = str_el_accessor.size(0);

    // Compute thread index corresponding in output tensor
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Declare variables
    scalar value = bottom;
    scalar sum;
    int idx_x, idx_y;

    // Compute the value of output[y][x]
    if (x < m && y < n) {
        for (int j = q-1; j >= 0; j--) {
            for (int i = p-1; i >= 0; i--) {
                if (footprint_accessor[j][i]) {
                    idx_x = x - (i - origin_x);
                    idx_y = y - (j - origin_y);
                    if (0 <= idx_x && idx_x < m && 0 <= idx_y && idx_y < n) {
                        sum = input_accessor[idx_y][idx_x] + str_el_accessor[j][i];
                        if (value < sum) value = sum;
                    }
                }
            }
        }
        output_accessor[y][x] = value;
    }
}

/* Implementations */
// Erosion
template <typename scalar>
torch::Tensor erosion(torch::Tensor input,
                      torch::Tensor str_el,
                      torch::Tensor footprint,
                      int origin_x,
                      int origin_y,
                      char border_type,
                      scalar top,
                      scalar bottom,
                      const int block_size_x,
                      const int block_size_y) {

    // Compute output size
    const auto m = input.size(1);
    const auto n = input.size(0);
    const auto p = str_el.size(1);
    const auto q = str_el.size(0);

    // Initialization
    auto options = torch::TensorOptions().device(input.device()).dtype(input.dtype());
    torch::Tensor output_tensor = torch::zeros({n, m}, options);

    // Switch between CPU and GPU
    if (input.is_cuda()) {
        /* GPU */
        // Create accessors
        auto input_accessor = input.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();
        auto str_el_accessor = str_el.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();
        auto footprint_accessor = footprint.packed_accessor32<bool, 2, torch::RestrictPtrTraits>();
        auto output_accessor = output_tensor.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();

        // Block & Grid parameters
        const int grid_x = ((m - 1) / block_size_x) + 1;
        const int grid_y = ((n - 1) / block_size_y) + 1;

        const dim3 block_size(block_size_x, block_size_y, 1);
        const dim3 grid_size(grid_x, grid_y, 1);

        // Launch of the kernel
        erosion_cuda_kernel<<<grid_size, block_size>>>(input_accessor, str_el_accessor, footprint_accessor,
                                                       output_accessor, origin_x, origin_y, border_type, top, bottom);
    } else {
        /* CPU */
        // Create accessors
        auto input_accessor = input.accessor<scalar, 2>();
        auto str_el_accessor = str_el.accessor<scalar, 2>();
        auto footprint_accessor = footprint.accessor<bool, 2>();
        auto output_accessor = output_tensor.accessor<scalar, 2>();

        scalar value;
        scalar difference;
        int idx_x, idx_y;
        // Computation
        for (int y = 0; y < n; y++) {
            for (int x = 0; x < m; x++) {
                value = top;
                // Compute the value of output[y][x]
                for (int j = 0; j < q; j++) {
                    for (int i = 0; i < p; i++) {
                        if (footprint_accessor[j][i]) {
                            idx_x = x + (i - origin_x);
                            idx_y = y + (j - origin_y);
                            if (0 <= idx_x && idx_x < m && 0 <= idx_y && idx_y < n) {
                                difference = input_accessor[idx_y][idx_x] - str_el_accessor[j][i];
                                if (value > difference) value = difference;
                            } else if (border_type == 'e') {
                                value = bottom;
                                goto end;
                            }
                        }
                    }
                }
                end: output_accessor[y][x] = value;
            }
        }
    }

    return output_tensor;
}

// Dilation
template <typename scalar>
torch::Tensor dilation(torch::Tensor input,
                       torch::Tensor str_el,
                       torch::Tensor footprint,
                       int origin_x,
                       int origin_y,
                       scalar bottom,
                       int block_size_x,
                       int block_size_y) {

    // Compute output size
    const auto m = input.size(1);
    const auto n = input.size(0);
    const auto p = str_el.size(1);
    const auto q = str_el.size(0);

    // Initialization
    auto options = torch::TensorOptions().device(input.device()).dtype(input.dtype());
    torch::Tensor output_tensor = torch::zeros({n, m}, options);

    // Switch between CPU and GPU
    if (input.is_cuda()) {
        /* GPU */
        // Create accessors
        auto input_accessor = input.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();
        auto str_el_accessor = str_el.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();
        auto footprint_accessor = footprint.packed_accessor32<bool, 2, torch::RestrictPtrTraits>();
        auto output_accessor = output_tensor.packed_accessor32<scalar, 2, torch::RestrictPtrTraits>();

        // Block & Grid parameters
        const int grid_x = ((m - 1) / block_size_x) + 1;
        const int grid_y = ((n - 1) / block_size_y) + 1;

        const dim3 block_size(block_size_x, block_size_y, 1);
        const dim3 grid_size(grid_x, grid_y, 1);

        // Launch of the kernel
        dilation_cuda_kernel<<<grid_size, block_size>>>(input_accessor, str_el_accessor, footprint_accessor,
                                                        output_accessor, origin_x, origin_y, bottom);
    } else {
        /* CPU */
        // Create accessors
        auto input_accessor = input.accessor<scalar, 2>();
        auto str_el_accessor = str_el.accessor<scalar, 2>();
        auto footprint_accessor = footprint.accessor<bool, 2>();
        auto output_accessor = output_tensor.accessor<scalar, 2>();

        scalar value;
        scalar sum;
        int idx_x, idx_y;
        // Computation
        for (int y = 0; y < n; y++) {
            for (int x = 0; x < m; x++) {
                value = bottom;
                // Compute the value of output[y][x]
                for (int j = q-1; j >= 0; j--) {
                    for (int i = p-1; i >= 0; i--) {
                        if (footprint_accessor[j][i]) {
                            idx_x = x - (i - origin_x);
                            idx_y = y - (j - origin_y);
                            if (0 <= idx_x && idx_x < m && 0 <= idx_y && idx_y < n) {
                                sum = input_accessor[idx_y][idx_x] + str_el_accessor[j][i];
                                if (value < sum) value = sum;
                            }
                        }
                    }
                }
                output_accessor[y][x] = value;
            }
        }
    }

    return output_tensor;
}
