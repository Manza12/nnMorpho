#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <float.h>
#include <assert.h>

#include <iostream>
#include <stdio.h>

// Macros
#define INF FLT_MAX


/* CUDA kernels */
__global__ void erosion_cuda_kernel(
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> input_tensor,
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> strel_tensor,
		torch::PackedTensorAccessor32<float,2> output_tensor) {
	
	/* Sizes */
	// Input
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	
	// Strel
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	// Output
	const auto output_width = output_tensor.size(0);
	const auto output_height = output_tensor.size(1);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Initialize temporal variables 
	float value = INF;
	float candidate;
	
	// Compute the value of output[y][x]
	if (x < output_width && y < output_height) {
		for (int j = 0; j < strel_height; j++) {
			for (int i = 0; i < strel_width; i++) {
				candidate = input_tensor[x + i][y + j] - strel_tensor[i][j];
				if (candidate < value) {
					value = candidate;
				}
			}
		}
		output_tensor[x][y] = value;
	}
}

__global__ void dilation_cuda_kernel(
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> input_tensor,
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> strel_tensor,
		torch::PackedTensorAccessor32<float,2> output_tensor) {
	
	/* Sizes */
	// Input
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	
	// Strel
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	// Output
	const auto output_width = output_tensor.size(0);
	const auto output_height = output_tensor.size(1);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Initialize temporal variables 
	float value = -INF;
	float candidate;
	
	// Compute the value of output[y][x]
	if (x < output_width && y < output_height) {
		for (int j = 0; j < strel_height; j++) {
			for (int i = 0; i < strel_width; i++) {
				candidate = input_tensor[x + i][y + j] + strel_tensor[strel_width - (i + 1)][strel_height - (j + 1)];
				if (candidate > value) {
					value = candidate;
				}
			}
		}
		output_tensor[x][y] = value;
	}
}

__global__ void erosion_batched_cuda_kernel(
		const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> input_tensor,
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> strel_tensor,
		torch::PackedTensorAccessor32<float,3> output_tensor) {
	
	/* Sizes */
	// Input
	const auto batch_size = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto input_width = input_tensor.size(2);
	
	// Strel
	const auto strel_height = strel_tensor.size(0);
	const auto strel_width = strel_tensor.size(1);
	
	// Output
	const auto output_height = output_tensor.size(1);
	const auto output_width = output_tensor.size(2);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Initialize temporal variables 
	float value;
	float candidate;
	
	// Compute the value of output[y][x]
	if (x < output_width && y < output_height) {
		for (int k = 0; k < batch_size; k++) {
			value = INF;
			for (int j = 0; j < strel_height; j++) {
				for (int i = 0; i < strel_width; i++) {
					candidate = input_tensor[k][y + j][x + i] - strel_tensor[j][i];
					if (candidate < value) {
						value = candidate;
					}
				}
			}
			output_tensor[k][y][x] = value;
		}
	}
}

__global__ void dilation_batched_cuda_kernel(
		const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> input_tensor,
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> strel_tensor,
		torch::PackedTensorAccessor32<float,3> output_tensor) {
	
	/* Sizes */
	// Input
	const auto batch_size = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto input_width = input_tensor.size(2);
	
	// Strel
	const auto strel_height = strel_tensor.size(0);
	const auto strel_width = strel_tensor.size(1);
	
	// Output
	const auto output_height = output_tensor.size(1);
	const auto output_width = output_tensor.size(2);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Initialize temporal variables 
	float value;
	float candidate;
	
	// Compute the value of output[y][x]
	if (x < output_width && y < output_height) {
		for (int k = 0; k < batch_size; k++) {
			value = -INF;
			for (int j = 0; j < strel_height; j++) {
				for (int i = 0; i < strel_width; i++) {
					candidate = input_tensor[k][y + j][x + i] + strel_tensor[strel_height - (j + 1)][strel_width - (i + 1)];
					if (candidate > value) {
						value = candidate;
					}
				}
			}
			output_tensor[k][y][x] = value;
		}
	}
}

__global__ void partial_erosion_cuda_kernel(
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> input_tensor,
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> strel_tensor,
		torch::PackedTensorAccessor32<float,2> output_tensor) {
	
	/* Sizes */
	// Input
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	
	// Strel
	const auto strel_height = strel_tensor.size(1);
	
	// Output
	const auto output_width = output_tensor.size(0);
	const auto output_height = output_tensor.size(1);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Initialize temporal variables 
	float value = INF;
	float candidate;
	
	// Compute the value of output[y][x]
	if (x < output_width && y < output_height) {		
		for (int j = 0; j < strel_height; j++) {
			candidate = input_tensor[x][y + j] - strel_tensor[x][j];
			if (candidate < value) {
				value = candidate;
			}
		}
		output_tensor[x][y] = value;
	}
}

__global__ void erosion_forward_cuda_kernel(
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> input_tensor,
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> strel_tensor,
		torch::PackedTensorAccessor32<float,2> output_tensor,
		torch::PackedTensorAccessor32<short,3> indexes) {
	
	/* Sizes */
	// Input
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	
	// Strel
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	// Output
	const auto output_width = output_tensor.size(0);
	const auto output_height = output_tensor.size(1);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Initialize temporal variables 
	float value = INF;
	float candidate;
	int index_i;
	int index_j;
	
	// Compute the value of output[y][x]
	if (x < output_width && y < output_height) {
		for (int i = 0; i < strel_width; i++) {
			for (int j = 0; j < strel_height; j++) {
				candidate = input_tensor[x + i][y + j] - strel_tensor[i][j];
				if (candidate < value) {
					value = candidate;
					index_i = i;
					index_j = j;
				}
			}
		}
		output_tensor[x][y] = value;
		indexes[x][y][0] = index_i;
		indexes[x][y][1] = index_j;
	}
}

__global__ void erosion_backward_cuda_kernel(
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> grad_output_accessor,
		const torch::PackedTensorAccessor32<short,3,torch::RestrictPtrTraits> indexes_accessor,
		torch::PackedTensorAccessor32<float,2> grad_input_accessor) {
	
	/* Sizes */
	// Grad Output
	const auto grad_output_width = grad_output_accessor.size(0);
	const auto grad_output_height = grad_output_accessor.size(1);
	
	// Indexes
	const auto indexes_width = indexes_accessor.size(0);
	const auto indexes_height = indexes_accessor.size(1);
	
	// Grad Input
	const auto grad_input_width = grad_input_accessor.size(0);
	const auto grad_input_height = grad_input_accessor.size(1);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Add the value to the grad_input_accessor
	if (x < grad_output_width && y < grad_output_height) {
		short index_i = indexes_accessor[x][y][0];
		short index_j = indexes_accessor[x][y][1];
		atomicAdd(&grad_input_accessor[index_i][index_j], -grad_output_accessor[x][y]);
	}
}

__global__ void dilation_forward_cuda_kernel(
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> input_tensor,
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> strel_tensor,
		torch::PackedTensorAccessor32<float,2> output_tensor,
		torch::PackedTensorAccessor32<short,3> indexes) {
	
	/* Sizes */
	// Input
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	
	// Strel
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	// Output
	const auto output_width = output_tensor.size(0);
	const auto output_height = output_tensor.size(1);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Initialize temporal variables 
	float value = -INF;
	float candidate;
	int index_i;
	int index_j;
	
	// Compute the value of output[y][x]
	if (x < output_width && y < output_height) {
		for (int j = 0; j < strel_height; j++) {
			for (int i = 0; i < strel_width; i++) {
				candidate = input_tensor[x + i][y + j] + strel_tensor[strel_width - (i + 1)][strel_height - (j + 1)];
				if (candidate > value) {
					value = candidate;
					index_i = strel_width - (i + 1);
					index_j = strel_height - (j + 1);
				}
			}
		}
		output_tensor[x][y] = value;
		indexes[x][y][0] = index_i;
		indexes[x][y][1] = index_j;
	}
}

__global__ void dilation_backward_cuda_kernel(
		const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> grad_output_accessor,
		const torch::PackedTensorAccessor32<short,3,torch::RestrictPtrTraits> indexes_accessor,
		torch::PackedTensorAccessor32<float,2> grad_input_accessor) {
	
	/* Sizes */
	// Grad Output
	const auto grad_output_width = grad_output_accessor.size(0);
	const auto grad_output_height = grad_output_accessor.size(1);
	
	// Indexes
	const auto indexes_width = indexes_accessor.size(0);
	const auto indexes_height = indexes_accessor.size(1);
	
	// Grad Input
	const auto grad_input_width = grad_input_accessor.size(0);
	const auto grad_input_height = grad_input_accessor.size(1);
	
	// Compute thread index corresponding in output tensor
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Add the value to the grad_input_accessor
	if (x < grad_output_width && y < grad_output_height) {
		short index_i = indexes_accessor[x][y][0];
		short index_j = indexes_accessor[x][y][1];
		atomicAdd(&grad_input_accessor[index_i][index_j], grad_output_accessor[x][y]);
	}
}

/* CUDA */
torch::Tensor erosion_cuda(
    torch::Tensor input_tensor,
    torch::Tensor strel_tensor,
    torch::Tensor block_shape) {

	// Compute output size
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	const auto output_width = input_width - strel_width + 1;
	const auto output_height = input_height - strel_height + 1;
  	
  	// Initialize output tensor
  	auto options = torch::TensorOptions().device(input_tensor.device());
  	torch::Tensor output_tensor = torch::zeros({output_width, output_height}, options);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_width = block_ptr[0];
  	const short block_height = block_ptr[1];
  	
	const int grid_width = ((output_width - 1) / block_width) + 1;
	const int grid_height = ((output_height - 1) / block_height) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto input_accessor = input_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto strel_accessor = strel_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto output_accessor = output_tensor.packed_accessor32<float,2>();

	// Launch of the kernel
	erosion_cuda_kernel<<<grid_size, block_size>>>(input_accessor, strel_accessor, output_accessor);
	
  	return output_tensor;
}

torch::Tensor dilation_cuda(
    torch::Tensor input_tensor,
    torch::Tensor strel_tensor,
    torch::Tensor block_shape) {

	// Compute output size
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	const auto output_width = input_width - strel_width + 1;
	const auto output_height = input_height - strel_height + 1;
  	
  	// Initialize output tensor
  	auto options = torch::TensorOptions().device(input_tensor.device());
  	torch::Tensor output_tensor = torch::zeros({output_width, output_height}, options);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_width = block_ptr[0];
  	const short block_height = block_ptr[1];
  	
	const int grid_width = ((output_width - 1) / block_width) + 1;
	const int grid_height = ((output_height - 1) / block_height) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto input_accessor = input_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto strel_accessor = strel_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto output_accessor = output_tensor.packed_accessor32<float,2>();

	// Launch of the kernel
	dilation_cuda_kernel<<<grid_size, block_size>>>(input_accessor, strel_accessor, output_accessor);
	
  	return output_tensor;
}

torch::Tensor erosion_batched_cuda(
    torch::Tensor input_tensor,
    torch::Tensor strel_tensor,
    torch::Tensor block_shape) {

	// Compute output size
	const auto batch_size = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto input_width = input_tensor.size(2);
	const auto strel_height = strel_tensor.size(0);
	const auto strel_width = strel_tensor.size(1);
	
	const auto output_height = input_height - strel_height + 1;
	const auto output_width = input_width - strel_width + 1;
  	
  	// Initialize output tensor
  	auto options = torch::TensorOptions().device(input_tensor.device());
  	torch::Tensor output_tensor = torch::zeros({batch_size, output_height, output_width}, options);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_height = block_ptr[0];
  	const short block_width = block_ptr[1];
  	
	const int grid_height = ((output_height - 1) / block_height) + 1;
	const int grid_width = ((output_width - 1) / block_width) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto input_accessor = input_tensor.packed_accessor32<float,3,torch::RestrictPtrTraits>();
	auto strel_accessor = strel_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto output_accessor = output_tensor.packed_accessor32<float,3>();

	// Launch of the kernel
	erosion_batched_cuda_kernel<<<grid_size, block_size>>>(input_accessor, strel_accessor, output_accessor);
	
  	return output_tensor;
}

torch::Tensor dilation_batched_cuda(
    torch::Tensor input_tensor,
    torch::Tensor strel_tensor,
    torch::Tensor block_shape) {

	// Compute output size
	const auto batch_size = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto input_width = input_tensor.size(2);
	const auto strel_height = strel_tensor.size(0);
	const auto strel_width = strel_tensor.size(1);
	
	const auto output_height = input_height - strel_height + 1;
	const auto output_width = input_width - strel_width + 1;
  	
  	// Initialize output tensor
  	auto options = torch::TensorOptions().device(input_tensor.device());
  	torch::Tensor output_tensor = torch::zeros({batch_size, output_height, output_width}, options);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_height = block_ptr[0];
  	const short block_width = block_ptr[1];
  	
	const int grid_height = ((output_height - 1) / block_height) + 1;
	const int grid_width = ((output_width - 1) / block_width) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto input_accessor = input_tensor.packed_accessor32<float,3,torch::RestrictPtrTraits>();
	auto strel_accessor = strel_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto output_accessor = output_tensor.packed_accessor32<float,3>();

	// Launch of the kernel
	dilation_batched_cuda_kernel<<<grid_size, block_size>>>(input_accessor, strel_accessor, output_accessor);
	
  	return output_tensor;
}

torch::Tensor partial_erosion_cuda(
    torch::Tensor input_tensor,
    torch::Tensor strel_tensor,
    torch::Tensor block_shape) {

	// Compute output size
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto strel_height = strel_tensor.size(1);

	const auto output_width = input_width;
	const auto output_height = input_height - strel_height + 1;
	
  	// Initialize output tensor
  	auto options = torch::TensorOptions().device(input_tensor.device());
  	torch::Tensor output_tensor = torch::zeros({output_width, output_height}, options);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_width = block_ptr[0];
  	const short block_height = block_ptr[1];
  	
	const int grid_width = ((output_width - 1) / block_width) + 1;
	const int grid_height = ((output_height - 1) / block_height) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto input_accessor = input_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto strel_accessor = strel_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto output_accessor = output_tensor.packed_accessor32<float,2>();

	// Launch of the kernel
	partial_erosion_cuda_kernel<<<grid_size, block_size>>>(input_accessor, strel_accessor, output_accessor);
	
  	return output_tensor;
}

std::vector<torch::Tensor> erosion_forward_cuda(
    torch::Tensor input_tensor,
    torch::Tensor strel_tensor,
    torch::Tensor block_shape) {

	// Compute output size
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	const auto output_width = input_width - strel_width + 1;
	const auto output_height = input_height - strel_height + 1;
  	
  	// Initialize output tensor
  	auto options_output = torch::TensorOptions().device(input_tensor.device());
  	torch::Tensor output_tensor = torch::zeros({output_width, output_height}, options_output);
  	
  	// Initialize indexes
  	auto options_indexes = torch::TensorOptions().device(input_tensor.device()).dtype(torch::kInt16);
  	torch::Tensor indexes = torch::zeros({output_width, output_height, 2}, options_indexes);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_width = block_ptr[0];
  	const short block_height = block_ptr[1];
  	
	const int grid_width = ((output_width - 1) / block_width) + 1;
	const int grid_height = ((output_height - 1) / block_height) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto input_accessor = input_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto strel_accessor = strel_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto output_accessor = output_tensor.packed_accessor32<float,2>();
	auto indexes_accessor = indexes.packed_accessor32<short,3>();

	// Launch of the kernel
	erosion_forward_cuda_kernel<<<grid_size, block_size>>>(input_accessor, strel_accessor, output_accessor, indexes_accessor);
	
  	return {output_tensor, indexes};
}


torch::Tensor erosion_backward_cuda(
    torch::Tensor grad_output,
    torch::Tensor indexes,
    torch::Tensor strel_shape,
    torch::Tensor block_shape) {

	// Compute output size
	const auto grad_output_width = grad_output.size(0);
	const auto grad_output_height = grad_output.size(1);
	
	// Compute Grad Input size
	short* strel_ptr = strel_shape.data_ptr<short>();
  	const short strel_width = strel_ptr[0];
  	const short strel_height = strel_ptr[1];
	
  	// Initialize output tensor
  	auto options = torch::TensorOptions().device(grad_output.device());
  	torch::Tensor grad_input = torch::zeros({strel_width, strel_height}, options);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_width = block_ptr[0];
  	const short block_height = block_ptr[1];
  	
	const int grid_width = ((grad_output_width - 1) / block_width) + 1;
	const int grid_height = ((grad_output_height - 1) / block_height) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto grad_output_accessor = grad_output.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto indexes_accessor = indexes.packed_accessor32<short,3,torch::RestrictPtrTraits>();
	auto grad_input_accessor = grad_input.packed_accessor32<float,2>();

	// Launch of the kernel
	erosion_backward_cuda_kernel<<<grid_size, block_size>>>(grad_output_accessor, indexes_accessor, grad_input_accessor);
	
  	return grad_input;
}

std::vector<torch::Tensor> dilation_forward_cuda(
    torch::Tensor input_tensor,
    torch::Tensor strel_tensor,
    torch::Tensor block_shape) {

	// Compute output size
	const auto input_width = input_tensor.size(0);
	const auto input_height = input_tensor.size(1);
	const auto strel_width = strel_tensor.size(0);
	const auto strel_height = strel_tensor.size(1);
	
	const auto output_width = input_width - strel_width + 1;
	const auto output_height = input_height - strel_height + 1;
  	
  	// Initialize output tensor
  	auto options_output = torch::TensorOptions().device(input_tensor.device());
  	torch::Tensor output_tensor = torch::zeros({output_width, output_height}, options_output);
  	
  	// Initialize indexes
  	auto options_indexes = torch::TensorOptions().device(input_tensor.device()).dtype(torch::kInt16);
  	torch::Tensor indexes = torch::zeros({output_width, output_height, 2}, options_indexes);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_width = block_ptr[0];
  	const short block_height = block_ptr[1];
  	
	const int grid_width = ((output_width - 1) / block_width) + 1;
	const int grid_height = ((output_height - 1) / block_height) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto input_accessor = input_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto strel_accessor = strel_tensor.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto output_accessor = output_tensor.packed_accessor32<float,2>();
	auto indexes_accessor = indexes.packed_accessor32<short,3>();

	// Launch of the kernel
	dilation_forward_cuda_kernel<<<grid_size, block_size>>>(input_accessor, strel_accessor, output_accessor, indexes_accessor);
	
  	return {output_tensor, indexes};
}


torch::Tensor dilation_backward_cuda(
    torch::Tensor grad_output,
    torch::Tensor indexes,
    torch::Tensor strel_shape,
    torch::Tensor block_shape) {

	// Compute output size
	const auto grad_output_width = grad_output.size(0);
	const auto grad_output_height = grad_output.size(1);
	
	// Compute Grad Input size
	short* strel_ptr = strel_shape.data_ptr<short>();
  	const short strel_width = strel_ptr[0];
  	const short strel_height = strel_ptr[1];
	
  	// Initialize output tensor
  	auto options = torch::TensorOptions().device(grad_output.device());
  	torch::Tensor grad_input = torch::zeros({strel_width, strel_height}, options);
  	
  	// Block & Grid parameters
  	short* block_ptr = block_shape.data_ptr<short>();
  	const short block_width = block_ptr[0];
  	const short block_height = block_ptr[1];
  	
	const int grid_width = ((grad_output_width - 1) / block_width) + 1;
	const int grid_height = ((grad_output_height - 1) / block_height) + 1;
	
	const dim3 block_size(block_width, block_height, 1);
	const dim3 grid_size(grid_width, grid_height, 1);

	// Create accessors
	auto grad_output_accessor = grad_output.packed_accessor32<float,2,torch::RestrictPtrTraits>();
	auto indexes_accessor = indexes.packed_accessor32<short,3,torch::RestrictPtrTraits>();
	auto grad_input_accessor = grad_input.packed_accessor32<float,2>();

	// Launch of the kernel
	dilation_backward_cuda_kernel<<<grid_size, block_size>>>(grad_output_accessor, indexes_accessor, grad_input_accessor);
	
  	return grad_input;
}
